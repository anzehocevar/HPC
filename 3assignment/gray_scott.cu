#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <omp.h>
#include <mpi.h>
#include "gray_scott.h"

// Helper macro to access 2D grid
#define IDX(i, j, size) ((i) * (size) + (j))

#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image.h"
#include "stb_image_write.h"


// visualize
void colormap(float value, unsigned char *r, unsigned char *g, unsigned char *b) {
    float x = fminf(fmaxf(value, 0.0f), 1.0f);
    *r = (unsigned char)(9*(1-x)*x*x*x*255);
    *g = (unsigned char)(15*(1-x)*(1-x)*x*x*255);
    *b = (unsigned char)(8.5*(1-x)*(1-x)*(1-x)*x*255);
}

void write_png(const char *filename, float *V, int size) {
    unsigned char *image = (unsigned char *)malloc(size * size * 3);

    float minV = V[0], maxV = V[0];
    for (int i = 1; i < size * size; i++) {
        if (V[i] < minV) minV = V[i];
        if (V[i] > maxV) maxV = V[i];
    }
    float range = maxV - minV;
    if (range < 1e-6f) range = 1.0f;

    for (int i = 0; i < size * size; i++) {
        float norm = (V[i] - minV) / range;
        unsigned char r, g, b;
        colormap(norm, &r, &g, &b);
        image[i * 3 + 0] = r;
        image[i * 3 + 1] = g;
        image[i * 3 + 2] = b;
    }

    stbi_write_png_compression_level = 9;
    stbi_write_png(filename, size, size, 3, image, size * 3);
    free(image);
}



// Reference function for initialization of U and V
void initUV2D(float *U, float *V, int size) {
    // Set initial values: U=1.0, V=0.0
    for (int i = 0; i < size; i++) {
        for (int j = 0; j < size; j++) {
            U[IDX(i, j, size)] = 1.0f;
            V[IDX(i, j, size)] = 0.0f;
        }
    }

    // Seed a small square in the center
    int r = size / 8;
    for (int i = size / 2 - r; i < size / 2 + r; i++) {
        for (int j = size / 2 - r; j < size / 2 + r; j++) {
            U[IDX(i, j, size)] = 0.75f;
            V[IDX(i, j, size)] = 0.25f;
        }
    }
}


double gray_scott2D(gs_config config){
    // Initialize vars from .h
    int size = config.n;
    int iterations = config.steps;
    float dt = config.dt;
    float du = config.du;
    float dv = config.dv;
    float f = config.f;
    float k = config.k;

    // Allocate memory
    float *U = (float *)malloc(size * size * sizeof(float));
    float *V = (float *)malloc(size * size * sizeof(float));
    float *U_next = (float *)malloc(size * size * sizeof(float));
    float *V_next = (float *)malloc(size * size * sizeof(float));

    
    // Initialize U and V
    initUV2D(U, V, size);

    /*
    YOUR SOLUTION GOES HERE
    Write a 2D Gray-Scott simulation in C/C++ using CUDA, OpenMPI, and OpenMP.
    */

    for(int it = 0;it < iterations; it++){
        // Update U and V using the Gray-Scott model
        #pragma omp parallel for collapse(2)
        for(int i = 0;i < size; i++){
            for(int j = 0;j < size;j++){
                // Get the indices of the neighbors
                int up = (i - 1 + size) % size;
                int down = (i + 1) % size;
                int left = (j - 1 + size) % size;
                int right = (j + 1) % size;

                // Compute the Laplacian
                float laplacian_U = U[IDX(up, j, size)] + U[IDX(down, j, size)] +
                                    U[IDX(i, left, size)] + U[IDX(i, right, size)] -
                                    4 * U[IDX(i, j, size)];

                float laplacian_V = V[IDX(up, j, size)] + V[IDX(down, j, size)] +
                                    V[IDX(i, left, size)] + V[IDX(i, right, size)] -
                                    4 * V[IDX(i, j, size)];

                // Update U and V
                U_next[IDX(i, j, size)] = U[IDX(i, j, size)] +
                                                dt * (du * laplacian_U - U[IDX(i, j, size)] * V[IDX(i, j, size)] * V[IDX(i, j, size)] +
                                                f * (1 - U[IDX(i, j, size)]));

                V_next[IDX(i, j, size)] = V[IDX(i, j, size)] +
                                                dt * (dv * laplacian_V + U[IDX(i,j,size)] * V[IDX(i,j,size)] * V[IDX(i,j,size)] -
                                                (f + k) * V[IDX(i,j,size)]);
            }
        }

        // Swap pointers
        float *temp = U;
        U = U_next;
        U_next = temp;

        temp = V;
        V = V_next;
        V_next = temp;
    }


    // return average concentartion of V
    double avgV = 0.0;
    for (int i = 0; i < size * size; i++) {
        avgV += V[i];
    }
    avgV /= (size * size);

    // Write output to file
    write_png("output.png", V, size);

    // Cleanup
    free(U);
    free(V);
    free(U_next);
    free(V_next);


    return avgV;
}


